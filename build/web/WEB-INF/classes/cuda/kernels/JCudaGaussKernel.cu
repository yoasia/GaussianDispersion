

#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<math.h>
extern "C"

__global__ void gauss(int n, double Q, double u, double wind_dir, int stability, double grid, double H,  double *result )
{

    double x, y, z; //Point coordinates
    double sigY, sigZ; //Sigmas
    double PI = 3.1415926535897;
    double distance;
    double windX, windY;
    double alpha;
    double downwind, crosswind;
    double a, b, c, d;
    double theta;
    double x_tmp;
    double y_tmp;
    double max;
    double translation_x = 0.0;
    double translation_y = 0.0;
    double wind_dir_deg = (double)(((90 - (int) wind_dir)) % 360);
    
    if(wind_dir_deg < 0)
        wind_dir_deg+= 360;

    //Number of thread
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iz = floor((double)(index/(n*n)));
    int iy = floor((double)(index%(n*n)/n));
    int ix = floor((double)(index%(n*n)%n));


    //calculate point coordinates in m
    x = (ix - (0.5*n))*grid;
    y = (iy - (0.5*n))*grid;
    z = iz * grid;


    x_tmp = x;
    y_tmp = y;

    //point rotation
    x = x_tmp*cos(-wind_dir*PI/180) - y_tmp*sin(-wind_dir*PI/180);
    y = x_tmp*sin(-wind_dir*PI/180) + y_tmp*cos(-wind_dir*PI/180);

    x_tmp = x;
    y_tmp = y;
    max = (n * grid)/2;

    //point translation
    if(wind_dir_deg <= 135 && wind_dir_deg >= 45){
        if(wind_dir_deg > 90){
            translation_x =  max * tan((wind_dir_deg - 90)*PI/180);
        }
        else{
            translation_x =  0.0 - max * tan((90 - wind_dir_deg ) * PI/180);
        }
         translation_y =  0.0 - max;
    }
    else if(wind_dir_deg < 225 && wind_dir_deg >= 135){
        if(wind_dir_deg > 180){
            translation_x = max;
             translation_y = max * tan((wind_dir_deg - 180)*PI/180);
        }
        else{
            translation_x = max ;
            translation_y = 0.0 - max * tan((180 - wind_dir_deg ) * PI/180);;
        }

    }
    else if(wind_dir_deg < 315 && wind_dir_deg >= 225){
        if(wind_dir_deg > 270){
            translation_x =  0.0 - max * tan((wind_dir_deg - 270 )*PI/180);
        }
        else{
            translation_x =  max * tan((270 - wind_dir_deg) * PI/180);
        }
         translation_y =  max;
    }
    else{
        if(wind_dir_deg < 90){
            translation_x =  0.0 - max;
            translation_y =  0.0 - max * tan((wind_dir_deg)*PI/180);;
        }
        else{
            translation_x = 0.0 - max;
             translation_y =  max * tan((360 - wind_dir_deg ) * PI/180);
        }
    }
    if(translation_x > 0){
            translation_x -= 1000;
    }
    else
        translation_x += 1000;
    
    if(translation_y > 0){
        translation_y -= 1000;
    }
    else
        translation_y += 1000;

    //move point
    x += translation_x;
    y += translation_y;


    int resultIndex = (iz * n * n) + (iy * n) + ix;

    //calculate wind x and wind y (it winds from wind_dir -180)
    windX = u * sin((wind_dir - 180)*PI/180);
    windY = u * cos((wind_dir - 180)*PI/180);

    //distance vector
    distance = sqrt(x*x + y*y);
    
    //calculate wind_dir_deg between wind vector and position vector
    alpha = acos((x*windX + y*windY)/(u*distance));

    //scalar projection
    downwind = distance * cos(alpha);
    crosswind = distance * sin(alpha);

    //Definition of parametrs a, b, c and d
    switch(stability){
        case 1:
            if(downwind < 100 & downwind > 0){
                a = 122.800;
                b = 0.94470;
            }
            else if(downwind >= 100 & downwind < 150){
                a = 158.080;
                b = 1.05420;
            }
            else if(downwind >= 100 & downwind < 0){
                a = 170.220;
                b = 1.09320;
            }
            else if(downwind >= 200 & downwind < 250){
                a = 179.520;
                b = 1.12620;
            }
            else if(downwind >= 250 & downwind < 300){
                a = 217.410;
                b = 1.26440;
            }
            else if(downwind >= 300 & downwind < 400){
                a = 258.89;
                b = 1.40940;
            }
            else if(downwind >= 400 & downwind < 500){
                a = 346.75;
                b = 1.7283;
            }
            else if(downwind >= 500 & downwind < 3110){
                a = 453.85;
                b = 2.1166;
            }
            else if(downwind >= 3110){
                a = 453.85;
                b = 2.1166;
            }

            c = 24.1670;
            d = 2.5334;

            break;
        case 2:
            // vertical
            if(downwind<200 && downwind>0){
                a=90.673;
                b=0.93198;
            }
            if(downwind>=200 && downwind<400){
                a=98.483;
                b=0.98332;
            }        
            if(downwind>=400){
                a=109.3;
                b=1.09710;
            }
            // cross wind
            c = 18.3330;
            d = 1.8096;
            break;
        case 3:
            // vertical
            a = 61.141;
            b = 0.91465;
            // cross wind
            c = 12.5;
            d = 1.0857;
            break;
        case 4:
            // vertical
            if(downwind<300 && downwind>0){
                a=34.459;
                b=0.86974;
            }
            if(downwind>=300 && downwind<1000){
                a=32.093;
                b=0.81066;
            }
            if(downwind>=1000 && downwind<3000){
                a=32.093;
                b=0.64403;
            }
            if(downwind>=3000 && downwind<10000){
                a=33.504;
                b=0.60486;
            }
            if(downwind>=10000 && downwind<30000){
                a=36.650;
                b=0.56589;
            }    
            if(downwind>=30000){
                a=44.053;
                b=0.51179;
            }
            // cross wind
            c = 8.3330;
            d = 0.72382;
            break;
        case 5:
            // vertical
            if(downwind<100 && downwind>0){
                a=24.26;
                b=0.83660;
            }
            if(downwind>=100 && downwind<300){
                a=23.331;
                b=0.81956;
            }
            if(downwind>=300 && downwind<1000){
                a=21.628;
                b=0.75660;
            }
            if(downwind>=1000 && downwind<2000){
                a=21.628;
                b=0.63077;
            }
            if(downwind>=2000 && downwind<4000){
                a=22.534;
                b=0.57154;
            }
            if(downwind>=4000 && downwind<10000){
                a=24.703;
                b=0.50527;
            }
            if(downwind>=10000 && downwind<20000){
                a=26.970;
                b=0.46713;
            }
            if(downwind>=20000 && downwind<40000){
                a=35.420;
                b=0.37615;
            }
            if(downwind>=40000){
                a=47.618;
                b=0.29592;
            }
            // cross wind
            c = 6.25;
            d = 0.54287;
            break;
        case 6:
            // vertical
            if(downwind<200 && downwind>0){
                a=15.209;
                b=0.81558;
            }
            if(downwind>=200 && downwind<700){
                a=14.457;
                b=0.78407;
            }
            if(downwind>=700 && downwind<1000){
                a=13.953;
                b=0.68465;
            }
            if(downwind>=1000 && downwind<2000){
                a=13.953;
                b=0.63227;
            }
            if(downwind>=2000 && downwind<3000){
                a=14.823;
                b=0.54503;
            }
            if(downwind>=3000 && downwind<7000){
                a=16.187;
                b=0.46490;
            }
            if(downwind>=7000 && downwind<15000){
                a=17.836;
                b=0.41507;
            }
            if(downwind>=15000 && downwind<30000){
                a=22.651;
                b=0.32681;
            }
            if(downwind>=30000 && downwind<60000){
                a=27.074;
                b=0.27436;
            }
            if(downwind>=60000){
                a=34.219;
                b=0.21716;
            }
            // cross wind
            c = 4.1667;
            d = 0.36191;
            break;
        default:
            break;
    }

    //calculate sigmaX and sigmaY
    sigZ=pow(a*(downwind/1000),b);
    if(sigZ > 5000)
        sigZ = 5000;
    
    theta=0.017453293*(c-d*log(downwind/1000));
    sigY=465.11628*downwind/1000*tan(theta);

    if (ix<n && iy<n && iz<n)
    {
        result[resultIndex] = Q/(2*u*PI*sigY*sigZ) * //1
        exp((-1.0)*pow(crosswind, 2)/(2.0*pow(sigY, 2))) * //2
        (exp((-1.0)*pow(z - H, 2)/(2.0*pow(sigZ, 2))) + //3a
        exp((-1.0)*pow(z + H, 2)/2.0*pow(sigZ, 2))) * 1000000; //3b
    }  

}





